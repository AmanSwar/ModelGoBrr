#include "ATen/core/TensorBody.h"
#include "c10/core/ScalarType.h"
#include "c10/util/BFloat16.h"

#include "c10/util/Half.h"
#include "rmsnorm.cuh"

#include <cassert>
#include <torch/extension.h>
extern "C" void launch_rmsnorm_fp16(const half *input_matrix,
                                               const half *weight_matrix,
                                               half *out_matrix, int M, int N,
                                               float eps) {
  launch_rmsnorm_fp16_vectorized(input_matrix, weight_matrix, out_matrix, M, N,
                                 eps);
}

torch::Tensor fused_rmsnorm(torch::Tensor input_matrix, torch::Tensor weight,
                            float eps = 1e-6f) {
  TORCH_CHECK(input_matrix.is_cuda(), "input must be CUDA");
  TORCH_CHECK(weight.is_cuda(), "weight must be CUDA");
  TORCH_CHECK(input_matrix.scalar_type() == at::kHalf,
              "input must be bfloat16");
  TORCH_CHECK(weight.scalar_type() == at::kHalf, "weight must be bfloat16");
  TORCH_CHECK(input_matrix.dim() == 3,
              "expected input shape (bs, seq_len, embed_dim)");

  // ensure contiguous
  if (!input_matrix.is_contiguous())
    input_matrix = input_matrix.contiguous();
  if (!weight.is_contiguous())
    weight = weight.contiguous();

  int64_t bs = input_matrix.size(0);
  int64_t seq_len = input_matrix.size(1);
  int64_t embed_dim = input_matrix.size(2);

  int M = static_cast<int>(bs * seq_len); // rows
  int N = static_cast<int>(embed_dim);    // columns

  TORCH_CHECK(weight.size(0) == N, "weight size must match embed dim");

  auto output = torch::empty_like(input_matrix);

  const half *in_ptr = reinterpret_cast<const half *>(
      input_matrix.data_ptr<at::Half>());
  const half *w_ptr =
      reinterpret_cast<const half *>(weight.data_ptr<at::Half>());
  half *out_ptr =
      reinterpret_cast<half *>(output.data_ptr<at::Half>());

  launch_rmsnorm_fp16(in_ptr, w_ptr, out_ptr, M, N, eps);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    TORCH_CHECK(false, "Kernel launch failed: ", hipGetErrorString(err));
  }

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  // m.def("rmsnorm_kernel_vec", &fused_rmsnorm, "Fused RMSNorm (BF16)");
  m.def("rmsnorm_kernel_vectorized", &fused_rmsnorm, "Fused RMSNorm (BF16)");
}